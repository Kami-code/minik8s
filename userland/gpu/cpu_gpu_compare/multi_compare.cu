#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

void init_matrix(int *matrix, int size){
	for (int i = 0; i < size; i++){
		matrix[i] = (int)(rand() & 0xff) / 66.6;
	}
}

void multi_cpu(int *d_matrixA, int *d_matrixB, int *d_matrixC, int m, int n, int o){
    for (int iy = 0; iy < m; iy++){
        for (int ix = 0; ix < o; ix++){
            int c = 0;
            for (int k = 0; k < n; k++){
                c += d_matrixA[iy * n + k] * d_matrixB[k * o + ix];
            }
            d_matrixC[iy * o + ix] = c;
        }
    }
}

__global__ void multi_gpu(int *d_matrixA, int *d_matrixB, int *d_matrixC, int m, int n, int o){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if(iy < m && ix < o) {
        int temp = 0;
        for(int i = 0; i < n; ++i){
            temp += d_matrixA[iy * n + i] * d_matrixB[i * o + ix];
        }
        d_matrixC[iy * o + ix] = temp;
        
    }
}

void print_result(int *matrixC, int *matrixCC, int x, int y){
	int *c = matrixC, *cc = matrixCC;
	for (int iy = 0; iy < y; iy++)
	{
		for (int ix = 0; ix < x; ix++)
		{
			printf("%d - %d = %d    ", c[ix], cc[ix],  c[ix]-cc[ix]);
		}
		c += x;
		cc += x;
		printf("\n");
	}
}

void matrix_multi(){
    int m = 1<<6;
	int n = 1<<10;
    int o = 1<<8;
    int *matrixA = (int *)malloc(sizeof(int) * m * n);
    int *matrixB = (int *)malloc(sizeof(int) * n * o);
    int *matrixC = (int *)malloc(sizeof(int) * m * o);
    int *matrixCC = (int *)malloc(sizeof(int) * m * o);
    init_matrix(matrixA, m * n);
    init_matrix(matrixB, n * o);
    clock_t cpuStart = clock();
    multi_cpu(matrixA, matrixB, matrixCC, m, n, o);
    clock_t cpuEnd = clock();
    float cpuTime = (float)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
	printf("cpu time:%f\n", cpuTime);
    int *d_matrixA, *d_matrixB, *d_matrixC;
    hipMalloc((void **)&d_matrixA, sizeof(int) * m * n);
    hipMalloc((void **)&d_matrixB, sizeof(int) * n * o);
    hipMalloc((void **)&d_matrixC, sizeof(int) * m * o);
    hipMemcpy(d_matrixA, matrixA, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, matrixB, sizeof(int) * n * o, hipMemcpyHostToDevice);
    int dimx = 32;
    int dimy = 32;
	dim3 block(dimx, dimy);
    dim3 grid(o / block.x + 1, m / block.y + 1);
    clock_t gpuStart = clock();
    multi_gpu<<<grid, block>>>(d_matrixA, d_matrixB, d_matrixC, m, n, o);
    clock_t gpuEnd = clock();
    float gpuTime = (float)(gpuEnd - gpuStart) / CLOCKS_PER_SEC;
	printf("gpu time:%f\n", gpuTime);
    hipMemcpy(matrixC, d_matrixC, sizeof(int) * m * o, hipMemcpyDeviceToHost);
    cout << "检验结果:" << endl;
    print_result(matrixC, matrixCC, o, m);
    free(matrixA);
    free(matrixB);
    free(matrixC);
    free(matrixCC);
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
}

int main(int argc, char *argv[])
{
    matrix_multi();
    return 0;
}