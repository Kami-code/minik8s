#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <time.h>

using namespace std;

void init_matrix(int *matrix, int size){
	for (int i = 0; i < size; i++){
		matrix[i] = (int)(rand() & 0xff) / 66.6;
	}
}

void add_cpu(int *matrixA, int *matrixB, int *matrixC, int x, int y){
	int *a = matrixA, *b = matrixB, *c = matrixC;
	for (int iy = 0; iy < y; iy++){
		for (int ix = 0; ix < x; ix++){
			c[ix] = a[ix] + b[ix];
		}
		a += x;
		b += x;
		c += x;
	}
}

__global__ void add_gpu(int *d_matrixA, int *d_matrixB, int *d_matrixC, int x, int y) {
    int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = iy * x + ix;
	if (ix < x && iy < y){
		d_matrixC[idx] = d_matrixA[idx] + d_matrixB[idx];
	}
}

void print_result(int *matrixC, int *matrixCC, int x, int y){
	int *c = matrixC, *cc = matrixCC;
	for (int iy = 0; iy < y; iy++)
	{
		for (int ix = 0; ix < x; ix++)
		{
			printf("%d - %d = %d    ", c[ix], cc[ix],  c[ix]-cc[ix]);
		}
		c += x;
		cc += x;
		printf("\n");
	}
	printf("\n");
}


void matrix_add_compare() {
    int m = 1<<14;
	int n = 1<<14;
    int *matrixA = (int *)malloc(sizeof(int) * m * n);
    int *matrixB = (int *)malloc(sizeof(int) * m * n);
    int *matrixC = (int *)malloc(sizeof(int) * m * n);
    int *matrixCC = (int *)malloc(sizeof(int) * m * n);
    init_matrix(matrixA, m * n);
    init_matrix(matrixB, m * n);
    int x = n, y = m;
    clock_t cpuStart = clock();
    add_cpu(matrixA, matrixB, matrixCC, x, y);
    clock_t cpuEnd = clock();
    float cpuTime = (float)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
	printf("cpu time:%f\n", cpuTime);
    int *d_matrixA, *d_matrixB, *d_matrixC;
    hipMalloc((void **)&d_matrixA, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixB, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixC, sizeof(int) * n * m);
    hipMemcpy(d_matrixA, matrixA, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, matrixB, sizeof(int) * n * m, hipMemcpyHostToDevice);

    int dimx = 32;
    int dimy = 32;
	dim3 block(dimx, dimy);
    dim3 grid(x / block.x + 1, y / block.y + 1);
    clock_t gpuStart = clock();
    add_gpu<<<grid, block>>>(d_matrixA, d_matrixB, d_matrixC, x, y);
    clock_t gpuEnd = clock();
    float gpuTime = (float)(gpuEnd - gpuStart) / CLOCKS_PER_SEC;
	printf("gpu time:%f\n", gpuTime);
    hipMemcpy(matrixC, d_matrixC, sizeof(int) * n * m, hipMemcpyDeviceToHost);
    // cout << "检验结果:" << endl;
    // print_result(matrixC, matrixCC, x, y);
    free(matrixA);
    free(matrixB);
    free(matrixC);
    free(matrixCC);
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
}

int main(int argc, char *argv[]) {
    matrix_add_compare();
    return 0;
}