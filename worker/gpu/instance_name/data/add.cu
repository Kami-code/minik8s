#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>

using namespace std;

__global__ void add_gpu(int *d_matrixA, int *d_matrixB, int *d_matrixC, int x, int y) {
    int ix = threadIdx.x + blockDim.x*blockIdx.x;
	int iy = threadIdx.y + blockDim.y*blockIdx.y;
	unsigned int idx = iy * x + ix;
	if (ix < x && iy < y){
		d_matrixC[idx] = d_matrixA[idx] + d_matrixB[idx];
	}
}




vector<vector<int>> matrix_add(vector<vector<int>> &a, vector<vector<int>> &b) {
    const int m = a.size(), n = a[0].size();
    int *matrixA = (int *)malloc(sizeof(int) * m * n);
    int *matrixB = (int *)malloc(sizeof(int) * m * n);
    int *matrixC = (int *)malloc(sizeof(int) * m * n);
    // cout << "矩阵输入:" << endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            matrixA[i * n + j] = a[i][j];
            matrixB[i * n + j] = b[i][j];
        }
        // for (int j = 0; j < n; j++) {
        //     cout << matrixA[i * n + j] << " ";
        // }
        // cout << "    ";
        // for (int j = 0; j < n; j++) {
        //     cout << matrixB[i * n + j] << " ";
        // }
        // cout << endl;
    }

    int *d_matrixA, *d_matrixB, *d_matrixC;
    hipMalloc((void **)&d_matrixA, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixB, sizeof(int) * n * m);
    hipMalloc((void **)&d_matrixC, sizeof(int) * n * m);
    hipMemcpy(d_matrixA, matrixA, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, matrixB, sizeof(int) * n * m, hipMemcpyHostToDevice);
    int x = n,y = m;
    int dimx = 32;
    int dimy = 32;
	dim3 block(dimx, dimy);
    dim3 grid(x / block.x + 1, y / block.y + 1);
    add_gpu<<<grid, block>>>(d_matrixA, d_matrixB, d_matrixC, x, y);
    hipMemcpy(matrixC, d_matrixC, sizeof(int) * n * m, hipMemcpyDeviceToHost);
    vector<int> temp(n, 0);
    vector<vector<int>> c(m, temp);
    // cout << "结果输出:" << endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            c[i][j] = matrixC[i * n + j];
            // cout << matrixC[i * n + j] << " ";
        }
        // cout << endl;
    }
    free(matrixA);
    free(matrixB);
    free(matrixC);
    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_matrixC);
    return c;
}

int main(int argc, char *argv[]) {
    vector<vector<int>> a{{1,2,3},{2,3,4},{3,4,5}}, b{{1,2,3},{4,5,6},{7,8,9}};
    vector<vector<int>> c = matrix_add(a, b);
    ofstream infile;
    infile.open("add.out");
    for (int i = 0; i < c.size(); i++){
        for (int j = 0; j < c[0].size(); j++){
            infile << c[i][j] << " ";
        }
        infile << "\n";
    }

    return 0;
}